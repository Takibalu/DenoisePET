#include "hip/hip_runtime.h"
#include "denoise.cuh"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <vector>


std::string to_string(DenoiseMethod method) {
    switch (method) {
    case IDENTITY:      return "identity";
    case BOX_FILTER:    return "box";
    case GAUSSIAN:      return "gaussian";
    case MEDIAN:        return "median";
    case BILATERAL:     return "bilateral";
    case NLM:           return "nlm";
    default:            return "unknown";
    }
}

__constant__ float kernel_3[3][3] = {
    {1, 2, 1},
    {2, 4, 2},
    {1, 2, 1}
};

__constant__ float kernel_5[5][5] = {
    {0.003663f, 0.014652f, 0.023173f, 0.014652f, 0.003663f},
    {0.014652f, 0.058608f, 0.092103f, 0.058608f, 0.014652f},
    {0.023173f, 0.092103f, 0.144448f, 0.092103f, 0.023173f},
    {0.014652f, 0.058608f, 0.092103f, 0.058608f, 0.014652f},
    {0.003663f, 0.014652f, 0.023173f, 0.014652f, 0.003663f}
};

__constant__ float kernel_9[9][9] = {
    {0.00000067, 0.00002292, 0.00019117, 0.00038771, 0.00024404, 0.00038771, 0.00019117, 0.00002292, 0.00000067},
    {0.00002292, 0.00078634, 0.00655965, 0.01331827, 0.00838894, 0.01331827, 0.00655965, 0.00078634, 0.00002292},
    {0.00019117, 0.00655965, 0.05472157, 0.11156508, 0.07025366, 0.11156508, 0.05472157, 0.00655965, 0.00019117},
    {0.00038771, 0.01331827, 0.11156508, 0.22749645, 0.14323822, 0.22749645, 0.11156508, 0.01331827, 0.00038771},
    {0.00024404, 0.00838894, 0.07025366, 0.14323822, 0.09037601, 0.14323822, 0.07025366, 0.00838894, 0.00024404},
    {0.00038771, 0.01331827, 0.11156508, 0.22749645, 0.14323822, 0.22749645, 0.11156508, 0.01331827, 0.00038771},
    {0.00019117, 0.00655965, 0.05472157, 0.11156508, 0.07025366, 0.11156508, 0.05472157, 0.00655965, 0.00019117},
    {0.00002292, 0.00078634, 0.00655965, 0.01331827, 0.00838894, 0.01331827, 0.00655965, 0.00078634, 0.00002292},
    {0.00000067, 0.00002292, 0.00019117, 0.00038771, 0.00024404, 0.00038771, 0.00019117, 0.00002292, 0.00000067}
};


__global__ void kernel_identity(const float* input, float* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int idx = y * width + x;
    output[idx] = input[idx]; //(no actual denoise)
}

__global__ void kernel_box_filter(const float* input, float* output, int width, int height, int window) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    float sum = 0.0f;
    int count = 0;

    for (int dy = -window; dy <= window; ++dy)
        for (int dx = -window; dx <= window; ++dx) {
            int nx = x + dx;
            int ny = y + dy;
            if (nx >= 0 && ny >= 0 && nx < width && ny < height) {
                sum += input[ny * width + nx];
                count++;
            }
        }

    output[y * width + x] = sum / count;
}

__global__ void kernel_gaussian_filter(const float* input, float* output, int width, int height, int window) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    float sum = 0.0f;
    float weightSum = 0.0f;

    for (int dy = -window; dy <= window; ++dy)
        for (int dx = -window; dx <= window; ++dx) {
            int nx = x + dx;
            int ny = y + dy;
            if (nx >= 0 && ny >= 0 && nx < width && ny < height) {
                float val = input[ny * width + nx];

                float weight;
                if (window == 1)
                    weight= kernel_3[dy + window][dx + window];
                if (window == 2)
                    weight= kernel_5[dy + window][dx + window];
                if (window == 4)
                    weight= kernel_9[dy + window][dx + window];
                sum += val * weight;
                weightSum += weight;
            }
        }

    output[y * width + x] = sum / weightSum;
}

__global__ void kernel_median_filter(const float* input, float* output, int width, int height, int window) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    //should be (window*2+1)*(window*2+1)
    float values[81];
    int count = 0;

    for (int dy = -window; dy <= window; ++dy)
        for (int dx = -window; dx <= window; ++dx) {
            int nx = x + dx;
            int ny = y + dy;
            if (nx >= 0 && ny >= 0 && nx < width && ny < height) {
                values[count++] = input[ny * width + nx];
            }
        }

    // Bubble sort 9 elements
    for (int i = 0; i < count - 1; ++i)
        for (int j = 0; j < count - i - 1; ++j)
            if (values[j] > values[j + 1]) {
                float tmp = values[j];
                values[j] = values[j + 1];
                values[j + 1] = tmp;
            }

    output[y * width + x] = values[count / 2];
}

__global__ void kernel_bilateral_filter(const float* input, float* output, int width, int height, float sigma_s, float sigma_r, int window) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    int idx = y * width + x;
    float center = input[idx];

    float sum = 0.0f;
    float weight_sum = 0.0f;

    for (int dy = -window; dy <= window; ++dy) {
        for (int dx = -window; dx <= window; ++dx) {
            int nx = x + dx;
            int ny = y + dy;

            if (nx >= 0 && ny >= 0 && nx < width && ny < height) {
                float neighbor = input[ny * width + nx];

                float spatial_dist2 = dx * dx + dy * dy;
                float intensity_diff = neighbor - center;
                float intensity_diff2 = intensity_diff * intensity_diff;

                float weight = expf(-spatial_dist2 / (2 * sigma_s * sigma_s) - intensity_diff2 / (2 * sigma_r * sigma_r));

                sum += neighbor * weight;
                weight_sum += weight;
            }
        }
    }

    output[idx] = sum / weight_sum;
}

__global__ void kernel_nlm_filter(const float* input, float* output, int width, int height, float h, int patch_radius,
                                  int search_radius)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int center_idx = y * width + x;

    float weight_sum = 0.0f;
    float result = 0.0f;

    for (int dy = -search_radius; dy <= search_radius; ++dy) {
        for (int dx = -search_radius; dx <= search_radius; ++dx) {
            int sx = x + dx;
            int sy = y + dy;

            if (sx < 0 || sy < 0 || sx >= width || sy >= height)
                continue;

            float dist2 = 0.0f;

            for (int py = -patch_radius; py <= patch_radius; ++py) {
                for (int px = -patch_radius; px <= patch_radius; ++px) {
                    int cx = x + px;
                    int cy = y + py;
                    int qx = sx + px;
                    int qy = sy + py;

                    if (cx >= 0 && cy >= 0 && cx < width && cy < height &&
                        qx >= 0 && qy >= 0 && qx < width && qy < height) {

                        float diff = input[cy * width + cx] - input[qy * width + qx];
                        dist2 += diff * diff;
                        }
                }
            }

            float weight = expf(-dist2 / (h * h));
            result += input[sy * width + sx] * weight;
            weight_sum += weight;
        }
    }

    output[center_idx] = result / (weight_sum + 1e-12f); //prevent div by 0
}

__global__ void kernel_identity_3D(const float* input, float* output, int width, int height, int depth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= width || y >= height || z >= depth) return;

    int idx = z * height * width + y * width + x;
    output[idx] = input[idx];
}

__global__ void kernel_box_filter_3D(const float* input, float* output, int width, int height, int depth, int window) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= width || y >= height || z >= depth) return;

    float sum = 0.0f;
    int count = 0;

    for (int dz = -window; dz <= window; ++dz) {
        for (int dy = -window; dy <= window; ++dy) {
            for (int dx = -window; dx <= window; ++dx) {
                int nx = x + dx;
                int ny = y + dy;
                int nz = z + dz;
                if (nx >= 0 && ny >= 0 && nz >= 0 && nx < width && ny < height && nz < depth) {
                    sum += input[nz * width * height + ny * width + nx];
                    count++;
                }
            }
        }
    }

    output[z * width * height + y * width + x] = sum / count;
}

__global__ void kernel_gaussian_filter_3D(const float* input, float* output,
                                          int width, int height, int depth, int window) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= width || y >= height || z >= depth) return;

    float sum = 0.0f;
    float weightSum = 0.0f;

    for (int dz = -window; dz <= window; ++dz) {
        for (int dy = -window; dy <= window; ++dy) {
            for (int dx = -window; dx <= window; ++dx) {
                int nx = x + dx;
                int ny = y + dy;
                int nz = z + dz;

                if (nx >= 0 && ny >= 0 && nz >= 0 && nx < width && ny < height && nz < depth) {
                    float val = input[nz * height * width + ny * width + nx];

                    // Compute 3D Gaussian weight
                    float distanceSq = dx * dx + dy * dy + dz * dz;
                    float sigma = float(window);  // adjust as needed
                    float weight = expf(-distanceSq / (2.0f * sigma * sigma));

                    sum += val * weight;
                    weightSum += weight;
                }
            }
        }
    }

    output[z * height * width + y * width + x] = sum / weightSum;
}


__global__ void kernel_median_filter_3D(const float* input, float* output, int width, int height, int depth, int window) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= width || y >= height || z >= depth) return;
    //should be (window*2+1)*(window*2+1)*(window*2+1)
    float values[729];
    int count = 0;

    for (int dz = -window; dz <= window; ++dz) {
        for (int dy = -window; dy <= window; ++dy) {
            for (int dx = -window; dx <= window; ++dx) {
                int nx = x + dx;
                int ny = y + dy;
                int nz = z + dz;

                if (nx >= 0 && ny >= 0 && nz >= 0 && nx < width && ny < height && nz < depth) {
                    values[count++] = input[nz * width * height + ny * width + nx];
                }
            }
        }
    }

    // Simple bubble sort for small count
    for (int i = 0; i < count - 1; ++i) {
        for (int j = 0; j < count - i - 1; ++j) {
            if (values[j] > values[j + 1]) {
                float tmp = values[j];
                values[j] = values[j + 1];
                values[j + 1] = tmp;
            }
        }
    }

    output[z * width * height + y * width + x] = values[count / 2];
}

__global__ void kernel_bilateral_filter_3D(const float* input, float* output,
                                           int width, int height, int depth,
                                           float sigma_s, float sigma_r, int window) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= width || y >= height || z >= depth) return;

    int center_idx = z * width * height + y * width + x;
    float center_val = input[center_idx];

    float sum = 0.0f;
    float weight_sum = 0.0f;

    for (int dz = -window; dz <= window; ++dz) {
        for (int dy = -window; dy <= window; ++dy) {
            for (int dx = -window; dx <= window; ++dx) {
                int nx = x + dx;
                int ny = y + dy;
                int nz = z + dz;

                if (nx >= 0 && ny >= 0 && nz >= 0 && nx < width && ny < height && nz < depth) {
                    int neighbor_idx = nz * width * height + ny * width + nx;
                    float neighbor_val = input[neighbor_idx];

                    float spatial_dist2 = dx * dx + dy * dy + dz * dz;
                    float intensity_diff2 = (neighbor_val - center_val) * (neighbor_val - center_val);

                    float weight = expf(-spatial_dist2 / (2.0f * sigma_s * sigma_s)
                                        - intensity_diff2 / (2.0f * sigma_r * sigma_r));

                    sum += neighbor_val * weight;
                    weight_sum += weight;
                }
            }
        }
    }

    output[center_idx] = sum / weight_sum;
}

__global__ void kernel_nlm_filter_3D(const float* input, float* output,
                                     int width, int height, int depth,
                                     float h, int search_radius, int patch_radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= width || y >= height || z >= depth) return;

    int center_idx = z * width * height + y * width + x;

    float weight_sum = 0.0f;
    float result = 0.0f;

    for (int dz = -search_radius; dz <= search_radius; ++dz) {
        for (int dy = -search_radius; dy <= search_radius; ++dy) {
            for (int dx = -search_radius; dx <= search_radius; ++dx) {
                int sx = x + dx;
                int sy = y + dy;
                int sz = z + dz;

                if (sx < 0 || sy < 0 || sz < 0 || sx >= width || sy >= height || sz >= depth)
                    continue;

                float dist2 = 0.0f;

                for (int pz = -patch_radius; pz <= patch_radius; ++pz) {
                    for (int py = -patch_radius; py <= patch_radius; ++py) {
                        for (int px = -patch_radius; px <= patch_radius; ++px) {
                            int cx = x + px, cy = y + py, cz = z + pz;
                            int qx = sx + px, qy = sy + py, qz = sz + pz;

                            if (cx < 0 || cy < 0 || cz < 0 || cx >= width || cy >= height || cz >= depth ||
                                qx < 0 || qy < 0 || qz < 0 || qx >= width || qy >= height || qz >= depth)
                                continue;

                            int c_idx = cz * width * height + cy * width + cx;
                            int q_idx = qz * width * height + qy * width + qx;

                            float diff = input[c_idx] - input[q_idx];
                            dist2 += diff * diff;
                        }
                    }
                }

                float weight = expf(-dist2 / (h * h));
                int s_idx = sz * width * height + sy * width + sx;
                result += input[s_idx] * weight;
                weight_sum += weight;
            }
        }
    }

    output[center_idx] = result / (weight_sum + 1e-12f); // prevent div by 0
}

void denoise(const float* input, float* output, int width, int height, DenoiseMethod method) {
    float *d_input, *d_output;
    size_t size = width * height * sizeof(float);

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((width + 15) / 16, (height + 15) / 16);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    switch (method) {
        case IDENTITY:
            kernel_identity<<<blocks, threads>>>(d_input, d_output, width, height);
            break;

        case BOX_FILTER:
            kernel_box_filter<<<blocks, threads>>>(d_input, d_output, width, height, window);
            break;

        case GAUSSIAN:
            kernel_gaussian_filter<<<blocks, threads>>>(d_input, d_output, width, height, window);
            break;

        case MEDIAN:
            kernel_median_filter<<<blocks, threads>>>(d_input, d_output, width, height, window);
            break;

        case BILATERAL:
            kernel_bilateral_filter<<<blocks, threads>>>(d_input, d_output, width, height, sigma_s, sigma_r, window);
            break;

        case NLM:
            kernel_nlm_filter<<<blocks, threads>>>(d_input, d_output, width, height, h, patch_radius, search_radius);
            break;

    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << std::endl;

    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
}


void denoise3D(const float* input, float* output, int width, int height, int depth, DenoiseMethod method) {
    float *d_input, *d_output;
    size_t size = width * height * depth * sizeof(float);

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    dim3 threads(8,8,8);
    dim3 blocks((width + 7) / 8, (height + 7) / 8, (depth + 7) / 8);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    switch (method) {
    case IDENTITY:
        kernel_identity_3D<<<blocks, threads>>>(d_input, d_output, width, height, depth);
        break;

    case BOX_FILTER:
        kernel_box_filter_3D<<<blocks, threads>>>(d_input, d_output, width, height, depth, window);
        break;

    case GAUSSIAN:
        kernel_gaussian_filter_3D<<<blocks, threads>>>(d_input, d_output, width, height, depth, window);
        break;

    case MEDIAN:
        kernel_median_filter_3D<<<blocks, threads>>>(d_input, d_output, width, height, depth, window);
        break;

    case BILATERAL:
        kernel_bilateral_filter_3D<<<blocks, threads>>>(d_input, d_output, width, height, depth, sigma_s, sigma_r, window);
        break;

    case NLM:
        kernel_nlm_filter_3D<<<blocks, threads>>>(d_input, d_output, width, height, depth, h, patch_radius, search_radius);
        break;

    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Denoise method " << to_string(method) << " took " << milliseconds << " ms" << std::endl;

    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
}
