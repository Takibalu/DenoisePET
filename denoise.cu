#include "hip/hip_runtime.h"
#include "denoise.cuh"
#include <hip/hip_runtime.h>

__global__ void kernel_denoise(const float* input, float* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int idx = y * width + x;
    output[idx] = input[idx]; // Identity (no actual denoise yet)
}

void denoise(const float* input, float* output, int width, int height) {
    float *d_input, *d_output;
    size_t size = width * height * sizeof(float);

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((width + 15) / 16, (height + 15) / 16);
    kernel_denoise<<<blocks, threads>>>(d_input, d_output, width, height);

    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
}
